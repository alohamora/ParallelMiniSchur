#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<hipsolver.h>
#include<hipsparse.h>
#include "sparse_matrix.h"
#include "mini_schur.h"
#define ll long long int

hipsparseMatDescr_t descrA;      
hipsolverSpHandle_t solver_handle;

void dense2SparseS(const float * __restrict__ d_A_dense, int **d_nnzPerVector, float **d_A, int **d_A_RowIndices, int **d_A_ColIndices, int &nnz, const hipsparseHandle_t handle, const int Nrows, const int Ncols);
void calculateInverse(float* y, float* d_y, float* x, sparse_matrix_t* D, sparse_matrix_t* U);
int solveLU(int N, int nnz, float* h_A, int* h_A_RowIndices, int* h_A_ColIndices, float* h_y, float* h_x);
void convert2cusparse(sparse_matrix_t* A);

__global__ void csr2dense(ll* rows, ll* cols, float* vals, ll n, float* dense){
    ll idx = rows[blockIdx.x] + threadIdx.x;
    if(idx < rows[blockIdx.x + 1])  dense[blockIdx.x + ((cols[idx]-1)*n)] = vals[idx];
}

void calculateMiniSchur(sparse_matrix_t* schur, sparse_matrix_t* D, sparse_matrix_t* L, sparse_matrix_t* U, sparse_matrix_t* G){
    hipsparseHandle_t handle;
    float *d_y = NULL, *y = NULL, *x = NULL, *d_x = NULL;
    int nnz = 0, *d_nnzPerVector;                        
    int *d_A_RowIndices, *d_A_ColIndices;
    float *d_A;
    x = (float *)malloc((U->n * U->m) * sizeof(float));
    y = (float *)malloc((U->n * U->m) * sizeof(float));
    // printf("%lld %lld\n", U->n, U->m);                         
    calculateInverse(y, d_y, x, D, U);
    // for(int i=0;i<(U->n * U->m);i++)    printf("%f %f\n", x[i], y[i]);
    hipMalloc((void **)&d_x, (D->n * U->m) * sizeof(float));
    hipMemcpy(d_x, x, (D->n * U->m) * sizeof(float), hipMemcpyHostToDevice);
    // dense2SparseS(d_x, &d_nnzPerVector, &d_A, &d_A_RowIndices, &d_A_ColIndices, nnz, handle, D->n, U->m);
}

void calculateInverse(float* y, float* d_y, float* x, sparse_matrix_t* D, sparse_matrix_t* U){
    sparse_matrix_t h_D;
    hipMalloc((void **)&(d_y), (D->n * U->m) * sizeof(float));
    hipMemset(d_y, 0.0f, (D->n * U->m) * sizeof(float));
    csr2dense<<<U->n, U->m>>>(U->rows, U->cols, U->vals, U->n, d_y);
    sparseMatrixCopy(D, &h_D, CPU);
    convert2cusparse(&h_D);
    hipDeviceSynchronize();
    hipMemcpy(y, d_y, (U->n * U->m) * sizeof(float), hipMemcpyDeviceToHost);
    for(int i=0;i<U->m;i++)
        solveLU(h_D.n, h_D.nnz, h_D.vals, h_D.irows, h_D.icols, y + (i*(U->n)), x + (i*(U->n)));
}

int solveLU(int N, int nnz, float* h_A, int* h_A_RowIndices, int* h_A_ColIndices, float* h_y, float* h_x){
    int singularity;
    cusolverSpScsrlsvluHost(solver_handle, N, nnz, descrA, h_A, h_A_RowIndices, h_A_ColIndices, h_y, 0.0000001, 0, h_x, &singularity);
    return singularity;
}

void convert2cusparse(sparse_matrix_t* A){
    A->irows = (int *)malloc((A->n+1) * sizeof(int));
    A->icols = (int *)malloc((A->nnz) * sizeof(int));
    for(int i=0;i<A->nnz;i++)  A->icols[i] = (int) A->cols[i];
    for(int i=0;i<=A->n;i++)  A->irows[i] = (int) A->rows[i] + 1;
}

void dense2SparseS(const float * __restrict__ d_A_dense, int **d_nnzPerVector, float **d_A, 
    int **d_A_RowIndices, int **d_A_ColIndices, int &nnz, const hipsparseHandle_t handle, const int Nrows, const int Ncols) {
    const int lda = Nrows;
    hipMalloc(&d_nnzPerVector[0], Nrows * sizeof(int));
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSnnz(handle, HIPSPARSE_DIRECTION_COLUMN, Nrows, Ncols, descrA, d_A_dense, lda, d_nnzPerVector[0], &nnz);
    hipMalloc(&d_A[0], nnz * sizeof(float));
    hipMalloc(&d_A_RowIndices[0], (Nrows + 1) * sizeof(int));
    hipMalloc(&d_A_ColIndices[0], nnz * sizeof(int));
    hipsparseSdense2csr(handle, Nrows, Ncols, descrA, d_A_dense, lda, d_nnzPerVector[0], d_A[0], d_A_RowIndices[0], d_A_ColIndices[0]);
}

void createHandles(){
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ONE);
    hipsolverSpCreate(&solver_handle);
}